#include "hip/hip_runtime.h"
#include "hitableList.h"
#include "operations.h"

__host__ __device__ void destroyObject(hitable* object) {
    if (object->next) {
        destroyObject(object->next);
    }
    object->destroy();
    delete object;
}

__host__ __device__ hitableList::~hitableList() {
    destroyObject(head);
}

__host__ __device__ void hitableList::addSingle(hitable* object) {
    if (head) {
        tail->next = object;
    } else {
        head = object;
        head->next = object;
    }
    tail = object;
}

__device__ bool hitableList::hit(const ray& r, float tMin, float tMax, hitRecord& rec) const {
    float depth = tMax;
    for (hitable* object = head; object; object = object->next) {
        if (object->hit(r, tMin, depth, rec)) {
            depth = rec.t;
        }
    }
    return depth != tMax;
}

hitableList* hitableList::create() {
    hitableList* list;
    checkCudaErrors(hipMalloc((void**)&list, sizeof(hitableList)));
    checkCudaErrors(hipGetLastError());
    return list;
}

void hitableList::destroy(hitableList* list) {
    checkCudaErrors(hipFree(list));
}

__global__ void addSingleInList(hitableList* list, hitable* object) {
    list->add(object);
}
