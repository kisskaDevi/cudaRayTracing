#include "hip/hip_runtime.h"
#include "sphere.h"

__device__ bool sphere::hit(const ray& r, float tMin, float tMax, hitRecord& rec) const {
    bool result = false;

    vec4 oc = r.getOrigin() - center;
    float a = dot(r.getDirection(), r.getDirection());
    float b = dot(oc, r.getDirection());
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - a * c;

    if (discriminant >= 0) {
        float temp = (-b - sqrt(discriminant)) / a;
        result = (temp < tMax&& temp > tMin);
        if (!result) {
            temp = (-b + sqrt(discriminant)) / a;
            result = (temp < tMax&& temp > tMin);
        }
        if (result) {
            rec.t = temp;
            rec.point = r.point(rec.t);
            rec.normal = (rec.point - center) / radius;
            rec.color = color;
            rec.mat = matptr;
        }
    }
    return result;
}