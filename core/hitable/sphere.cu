#include "hip/hip_runtime.h"
#include "sphere.h"

__device__ bool sphere::hit(const ray& r, float tMin, float tMax, hitRecord& rec) const {
    bool result = false;

    vec4 oc = r.getOrigin() - center;
    float a = dot(r.getDirection(), r.getDirection());
    float b = dot(oc, r.getDirection()) / a;
    float c = dot(oc, oc) - radius * radius / a;
    float discriminant = b * b - c;

    if (discriminant >= 0) {
        discriminant = sqrt(discriminant);
        float temp = -b - discriminant;
        result = (temp < tMax && temp > tMin);
        if (!result) {
            temp = -b + discriminant;
            result = (temp < tMax && temp > tMin);
        }
        if (result) {
            rec.t = temp;
            rec.point = r.point(rec.t);
            rec.normal = (rec.point - center) / radius;
            rec.color = color;
            rec.props = props;
            rec.mat = matptr;
        }
    }
    return result;
}

__global__ void createSphere(sphere** sph, vec4 cen, float r, vec4 color, const properties props, material* matptr) {
    *sph = new sphere(cen, r, color, props, matptr);
}